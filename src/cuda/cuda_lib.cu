#include "hip/hip_runtime.h"
#include "../../include/cuda/cuda_lib.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void matrix_mult (float* A, float* B, float* C, int a_row, int a_col, int b_row, int b_col) {
    // share memory 缓存A和B中对应的一对子矩阵，大小为BLOCK_SIZE * BLOCK_SIZE
    __shared__ float A_sub[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ float B_sub[BLOCK_SIZE * BLOCK_SIZE];
    // 获取当前线程所在的block和thread的id
    int block_id_row = blockIdx.x;
    int block_id_col = blockIdx.y;
    int thread_id_row = threadIdx.x;
    int thread_id_col = threadIdx.y;
    // 计算当前线程对应A矩阵的行号和B矩阵的列号，也就是C矩阵的行号和列号
    int c_row_id = block_id_row * BLOCK_SIZE + thread_id_row;
    int c_col_id = block_id_col * BLOCK_SIZE + thread_id_col;
    
    int sbmtx_begin = 0;
    float c = 0.0;
    float compensation = 0.0;
    for (sbmtx_begin = 0; sbmtx_begin < a_col; sbmtx_begin += BLOCK_SIZE) {// 遍历每一对A，B矩阵c_row_id，c_col_id所在行列的子区间
        // 当前线程加载A，B矩阵中对应子矩阵的指定元素，保证当前block中的线程同时加载完一对A，B子矩阵
        A_sub[thread_id_row * BLOCK_SIZE + thread_id_col] = (c_row_id < a_row && sbmtx_begin + thread_id_col < a_col) ? A[c_row_id * a_col + sbmtx_begin + thread_id_col] : 0;
        B_sub[thread_id_row * BLOCK_SIZE + thread_id_col] = (c_col_id < b_col && sbmtx_begin + thread_id_row < b_row) ? B[(sbmtx_begin + thread_id_row) * b_col + c_col_id] : 0;
        // 等待同一个block中的线程加载完毕
        __syncthreads ();
        // 计算A矩阵c_row_id行和B矩阵c_col_id列一个区间的内积，并将每个区间结果累计
        #pragma unroll
        for (int i = 0; i < BLOCK_SIZE; ++i) {
            // c += A_sub[thread_id_row * BLOCK_SIZE + i] * B_sub[i * BLOCK_SIZE + thread_id_col];
            // Kahan's Summation Formula
            float y = A_sub[thread_id_row * BLOCK_SIZE + i] * B_sub[i * BLOCK_SIZE + thread_id_col] - compensation;
            float t = c + y;// 发生舍入
            compensation = (t - c) - y;// 记录下舍入误差
            c = t;
        }
        __syncthreads ();
    }
    if (c_row_id < a_row && c_col_id < b_col) {
        C[c_row_id * b_col + c_col_id] = c;
    }
}
void cuda_matrix_mult (float* A, float* B, float* C, int a_row, int a_col, int b_row, int b_col) {// A*B=C
    int size_a = a_row * a_col;
    int size_b = b_row * b_col;
    int size_c = a_row * b_col;
    // 在显存上分配空间
    float* dev_A, *dev_B, *dev_C;
    hipMalloc ((void**) &dev_A, sizeof (float) * size_a);
    hipMalloc ((void**) &dev_B, sizeof (float) * size_b);
    hipMalloc ((void**) &dev_C, sizeof (float) * size_c);
    // copy数据到显存
    hipMemcpy (dev_A, A, sizeof (float) * size_a, hipMemcpyHostToDevice);
    hipMemcpy (dev_B, B, sizeof (float) * size_b, hipMemcpyHostToDevice);
    // 把结果C矩阵分割成grid_row * grid_col个BLOCK_SIZE * BLOCK_SIZE尺寸的block，可以认为C矩阵对应一个Grid
    int grid_row = a_row / BLOCK_SIZE + (a_row % BLOCK_SIZE == 0 ? 0 : 1);
    int grid_col = b_col / BLOCK_SIZE + (b_col % BLOCK_SIZE == 0 ? 0 : 1);
    dim3 grid (grid_row, grid_col);
    dim3 block (BLOCK_SIZE, BLOCK_SIZE);
    // 运行kernal函数
    matrix_mult <<<grid, block>>> (dev_A, dev_B, dev_C, a_row, a_col, b_row, b_col);
    // 把显存数据copy回内存
    hipMemcpy (C, dev_C, sizeof (float) * size_c, hipMemcpyDeviceToHost);
    // 释放显存
    hipFree (dev_A);
    hipFree (dev_B);
    hipFree (dev_C);
}

__global__ void tensor_add (float* A, float* B, float* C, int size) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int begin_idx = block_id * BLOCK_SIZE + thread_id;
    int read_offset = GRID_SIZE * BLOCK_SIZE;
    for (int i = begin_idx; i < size; i += read_offset) {// 这种方式尽可能保证显存数据的连续读取
        C[i] = A[i] + B[i];
    }
}
void cuda_tensor_add (float* A, float* B, float* C, int size) {
    float* dev_A, *dev_B, *dev_C;
    hipMalloc ((void**) &dev_A, sizeof (float) * size);
    hipMalloc ((void**) &dev_B, sizeof (float) * size);
    hipMalloc ((void**) &dev_C, sizeof (float) * size);

    hipMemcpy (dev_A, A, sizeof (float) * size, hipMemcpyHostToDevice);
    hipMemcpy (dev_B, B, sizeof (float) * size, hipMemcpyHostToDevice);
    tensor_add <<<GRID_SIZE, BLOCK_SIZE>>> (dev_A, dev_B, dev_C, size);
    hipMemcpy (C, dev_C, sizeof (float) * size, hipMemcpyDeviceToHost);
    // 释放显存
    hipFree (dev_A);
    hipFree (dev_B);
    hipFree (dev_C);
}

__global__ void scalar_tensor_mult (float* A, float* result, float s, int size) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int begin_idx = block_id * BLOCK_SIZE + thread_id;
    int read_offset = GRID_SIZE * BLOCK_SIZE;
    for (int i = begin_idx; i < size; i += read_offset) {// 这种方式尽可能保证显存数据的连续读取
        result[i] = A[i] * s;
    }
}
void cuda_scalar_tensor_mult (float* A, float* result, float s, int size) {
    float* dev_A, *dev_result;
    hipMalloc ((void**) &dev_A, sizeof (float) * size);
    hipMalloc ((void**) &dev_result, sizeof (float) * size);

    hipMemcpy (dev_A, A, sizeof (float) * size, hipMemcpyHostToDevice);
    scalar_tensor_mult <<<GRID_SIZE, BLOCK_SIZE>>> (dev_A, dev_result, s, size);
    hipMemcpy (result, dev_result, sizeof (float) * size, hipMemcpyDeviceToHost);
    // 释放显存
    hipFree (dev_A);
    hipFree (dev_result);
}

__global__ void element_abs_sum (float* A, int size, float* results) {
    __shared__ float sub_results[BLOCK_SIZE];
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;

    int begin_idx = block_id * BLOCK_SIZE + thread_id;
    int read_offset = GRID_SIZE * BLOCK_SIZE;
    if (begin_idx >= size) {
        sub_results[thread_id] = 0;
    } else {
        float r = 0;
        for (int i = begin_idx; i < size; i += read_offset) {
            r += fabs(A[i]);
        }
        sub_results[thread_id] = r;
    }
    // 将同一个block中得到的结果汇总到global存储中的results中
    __syncthreads ();
    int merge_offset = 1;
    int mask = 2;
    while (merge_offset <= BLOCK_SIZE) {
        if (thread_id % mask == 0 && thread_id + merge_offset < BLOCK_SIZE) {
            sub_results[thread_id] += sub_results[thread_id + merge_offset];
        }
        merge_offset = merge_offset * 2;
        mask = mask * 2;
        __syncthreads ();
    }
    if (thread_id == 0) {
        results[block_id] = sub_results[0];
    }
}
float cuda_element_abs_sum (float* A, int size) {
    float* results = (float*) malloc (sizeof (float) * GRID_SIZE);
    float* dev_A;
    float* dev_results;
    hipMalloc ((void**) &dev_A, sizeof (float) * size);
    hipMalloc ((void**) &dev_results, sizeof (float) * GRID_SIZE);

    hipMemcpy (dev_A, A, sizeof (float) * size, hipMemcpyHostToDevice);

    // 运行kernal函数
    element_abs_sum <<<GRID_SIZE, BLOCK_SIZE>>> (dev_A, size, dev_results);

    hipMemcpy (results, dev_results, sizeof (float) * GRID_SIZE, hipMemcpyDeviceToHost);
    hipFree (dev_results);
    hipFree (dev_A);
    float abs_sum = 0;
    // 在cpu端将显卡传回的数据汇总
    for (int i = 0; i < GRID_SIZE; ++i) {
        abs_sum += results[i];
    }
    free (results);
    return abs_sum;
}

__global__ void element_square_sum (float* A, int size, float* results) {
    __shared__ float sub_results[BLOCK_SIZE];
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    
    int begin_idx = block_id * BLOCK_SIZE + thread_id;
    int read_offset = GRID_SIZE * BLOCK_SIZE;
    if (begin_idx >= size) {
        sub_results[thread_id] = 0;
    } else {
        float r = 0;
        for (int i = begin_idx; i < size; i += read_offset) {
            r += A[i] * A[i];
        }
        sub_results[thread_id] = r;
    }
    // 将同一个block中得到的结果汇总到global存储中的results中
    __syncthreads ();
    int merge_offset = 1;
    int mask = 2;
    while (merge_offset <= BLOCK_SIZE) {
        if (thread_id % mask == 0 && thread_id + merge_offset < BLOCK_SIZE) {
            sub_results[thread_id] += sub_results[thread_id + merge_offset];
        }
        merge_offset = merge_offset * 2;
        mask = mask * 2;
        __syncthreads ();
    }
    if (thread_id == 0) {
        results[block_id] = sub_results[0];
    }
}
float cuda_element_square_sum (float* A, int size) {
    float* results = (float*) malloc (sizeof (float) * GRID_SIZE);
    float* dev_A;
    float* dev_results;
    hipMalloc ((void**) &dev_A, sizeof (float) * size);
    hipMalloc ((void**) &dev_results, sizeof (float) * GRID_SIZE);

    hipMemcpy (dev_A, A, sizeof (float) * size, hipMemcpyHostToDevice);
    
    // 运行kernal函数
    element_square_sum <<<GRID_SIZE, BLOCK_SIZE>>> (dev_A, size, dev_results);

    hipMemcpy (results, dev_results, sizeof (float) * GRID_SIZE, hipMemcpyDeviceToHost);
    hipFree (dev_results);
    hipFree (dev_A);
    float square_sum = 0;
    // 在cpu端将显卡传回的数据汇总
    for (int i = 0; i < GRID_SIZE; ++i) {
        square_sum += results[i];
    }
    free (results);
    return square_sum;
}

__global__ void element_square (float* A, int size) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int begin_idx = block_id * BLOCK_SIZE + thread_id;
    int read_offset = GRID_SIZE * BLOCK_SIZE;
    for (int i = begin_idx; i < size; i += read_offset) {
        A[i] = A[i] * A[i];
    }
}
void cuda_element_square (float* A, int size) {
    float* dev_A;
    hipMalloc ((void**) &dev_A, sizeof (float) * size);

    hipMemcpy (dev_A, A, sizeof (float) * size, hipMemcpyHostToDevice);
    element_square <<<GRID_SIZE, BLOCK_SIZE>>> (dev_A, size);
    hipMemcpy (A, dev_A, sizeof (float) * size, hipMemcpyDeviceToHost);
    // 释放显存
    hipFree (dev_A);
}

__global__ void element_mult (float* A, float* B, float* C, int size) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int begin_idx = block_id * BLOCK_SIZE + thread_id;
    int read_offset = GRID_SIZE * BLOCK_SIZE;
    for (int i = begin_idx; i < size; i += read_offset) {
        C[i] = A[i] * B[i];
    }
}
void cuda_element_mult (float* A, float* B, float* C, int size) {
    float* dev_A, *dev_B, *dev_C;
    hipMalloc ((void**) &dev_A, sizeof (float) * size);
    hipMalloc ((void**) &dev_B, sizeof (float) * size);
    hipMalloc ((void**) &dev_C, sizeof (float) * size);

    hipMemcpy (dev_A, A, sizeof (float) * size, hipMemcpyHostToDevice);
    hipMemcpy (dev_B, B, sizeof (float) * size, hipMemcpyHostToDevice);
    element_mult <<<GRID_SIZE, BLOCK_SIZE>>> (dev_A, dev_B, dev_C, size);
    hipMemcpy (C, dev_C, sizeof (float) * size, hipMemcpyDeviceToHost);
    // 释放显存
    hipFree (dev_A);
    hipFree (dev_B);
    hipFree (dev_C);
}
