#include "hip/hip_runtime.h"
#include "../../include/cuda/cuda_lib.h"
#include <stdlib.h>
#include <stdio.h>

__global__ void matrix_mult (float* A, float* B, float* C, int a_row, int a_col, int b_row, int b_col) {
    printf ("dddddddd");
    // share memory 缓存A和B中对应的一对子矩阵，大小为BLOCK_SIZE * BLOCK_SIZE
    __shared__ float A_sub[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ float B_sub[BLOCK_SIZE * BLOCK_SIZE];
    // 获取当前线程所在的block和thread的id
    int block_id_row = blockIdx.x;
    int block_id_col = blockIdx.y;
    int thread_id_row = threadIdx.x;
    int thread_id_col = threadIdx.y;
    // 计算当前线程对应A矩阵的行号和B矩阵的列号，也就是C矩阵的行号和列号
    int c_row_id = block_id_row * BLOCK_SIZE + thread_id_row;
    int c_col_id = block_id_col * BLOCK_SIZE + thread_id_col;
    
    int sbmtx_begin = 0;
    float c = 0.0;
    for (sbmtx_begin = 0; sbmtx_begin < a_col; sbmtx_begin += BLOCK_SIZE) {// 遍历每一对A，B矩阵c_row_id，c_col_id所在行列的子区间
        // 当前线程加载A，B矩阵中对应子矩阵的指定元素，保证当前block中的线程同时加载完一对A，B子矩阵
        A_sub[thread_id_row * BLOCK_SIZE + thread_id_col] = (c_row_id < a_row && sbmtx_begin + thread_id_col < a_col) ? A[c_row_id * a_col + sbmtx_begin + thread_id_col] : 0;
        B_sub[thread_id_row * BLOCK_SIZE + thread_id_col] = (c_col_id < b_col && sbmtx_begin + thread_id_row < b_row) ? B[(sbmtx_begin + thread_id_row) * b_col + c_col_id] : 0;
        // 等待同一个block中的线程加载完毕
        __syncthreads ();
        // 计算A矩阵c_row_id行和B矩阵c_col_id列一个区间的内积，并将每个区间结果累计
        #pragma unroll
        for (int i = 0; i < BLOCK_SIZE; ++i) {
            c += A_sub[thread_id_row * BLOCK_SIZE + i] * B_sub[i * BLOCK_SIZE + thread_id_col];
        }
        __syncthreads ();
    }
    if (c_row_id < a_row && c_col_id < b_col) {
        C[c_row_id * b_col + c_col_id] = c;
    }
}
void cuda_matrix_mult (float* A, float* B, float* C, int a_row, int a_col, int b_row, int b_col) {// A*B=C
    int size_a = a_row * a_col;
    int size_b = b_row * b_col;
    int size_c = a_row * b_col;
    // 在显存上分配空间
    float* dev_A, *dev_B, *dev_C;
    hipMalloc ((void**) &dev_A, sizeof (float) * size_a);
    hipMalloc ((void**) &dev_B, sizeof (float) * size_b);
    hipMalloc ((void**) &dev_C, sizeof (float) * size_c);
    // copy数据到显存
    hipMemcpy (dev_A, A, sizeof (float) * size_a, hipMemcpyHostToDevice);
    hipMemcpy (dev_B, B, sizeof (float) * size_b, hipMemcpyHostToDevice);
    // 把结果C矩阵分割成grid_row * grid_col个BLOCK_SIZE * BLOCK_SIZE尺寸的block，可以认为C矩阵对应一个Grid
    int grid_row = a_row / BLOCK_SIZE + (a_row % BLOCK_SIZE == 0 ? 0 : 1);
    int grid_col = b_col / BLOCK_SIZE + (b_col % BLOCK_SIZE == 0 ? 0 : 1);
    dim3 grid (grid_row, grid_col);
    dim3 block (BLOCK_SIZE, BLOCK_SIZE);
    // 运行kernal函数
    matrix_mult <<<grid, block>>> (dev_A, dev_B, dev_C, a_row, a_col, b_row, b_col);
    // 把显存数据copy回内存
    hipMemcpy (C, dev_C, sizeof (float) * size_c, hipMemcpyDeviceToHost);
    // 释放显存
    hipFree (dev_A);
    hipFree (dev_B);
    hipFree (dev_C);
}

void cuda_tensor_add (float* A, float* B, float* C, int size) {
}

void cuda_scalar_tensor_mult (float* A, float s, int size) {
}


__global__ void element_square_sum (float* A, int size, float* results) {
    printf ("ddddddddddd");
    __shared__ float sub_results[BLOCK_SIZE];
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    
    int begin_idx = block_id * BLOCK_SIZE + thread_id;
    int read_offset = GRID_SIZE * BLOCK_SIZE;
    if (begin_idx >= size) {
        sub_results[begin_idx] = 0;
    } else {
        float r = 0;
        for (int i = begin_idx; i < size; i += read_offset) {
            r += A[i] * A[i];
            printf ("%f ", r);
        }
        sub_results[begin_idx] = r;
    }
    // 将同一个block中得到的结果汇总到global存储中的results中
    __syncthreads ();
    int merge_offset = 1;
    int mask = 2;
    while (merge_offset <= BLOCK_SIZE) {
        if (thread_id % mask == 0 && thread_id + merge_offset < BLOCK_SIZE) {
            sub_results[thread_id] += sub_results[thread_id + merge_offset];
        }
        merge_offset = merge_offset * 2;
        mask = mask * 2;
        __syncthreads ();
    }
    if (thread_id == 0) {
        results[block_id] = sub_results[0];
    }
}
float cuda_element_square_sum (float* A, int size) {
    float* results = (float*) malloc (sizeof (float) * GRID_SIZE);
    float* dev_A;
    float* dev_results;
    hipMalloc ((void**) &dev_A, sizeof (float) * size);
    hipMalloc ((void**) &dev_results, sizeof (float) * GRID_SIZE);

    hipMemcpy (dev_A, A, sizeof (float) * size, hipMemcpyHostToDevice);
    
    // 运行kernal函数
    element_square_sum <<<GRID_SIZE, BLOCK_SIZE>>> (dev_A, size, dev_results);

    hipMemcpy (A, dev_A, sizeof (float) * size, hipMemcpyDeviceToHost);
    hipMemcpy (results, dev_results, sizeof (float) * size, hipMemcpyDeviceToHost);

    hipFree (dev_results);
    hipFree (dev_A);
    float square_sum = 0;
    // 在cpu端将显卡传回的数据汇总
    for (int i = 0; i < GRID_SIZE; ++i) {
        square_sum += results[i];
    }
    free (results);
    return square_sum;
}

void cuda_element_square (float* A, int size) {
}

void cuda_element_mult (float* A, float* B, int size) {
}
